#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#define M_ 2  
#define N_ 4 
#define P_ 3 

#define MAX 100
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

#define show(matrix, lenm, lenn) for (r = 0; r < lenm; r++){for (c = 0; c < lenn; c++){printf("%.6f ", matrix[r*lenn+c]);}printf("\n");}printf("\n");

__global__ void InitMatrix(float **m, unsigned int rows, unsigned int cols, int seed)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    hiprand_init((row*cols + col) * seed, /* the seed controls the sequence of random values that are produced */
              row, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);
    if (row < rows && col < cols)  
    {  
        //m[row][col] = curand_uniform(&state);
        m[row][col] = 1;
    }  
}  

__global__ void Multiply(float **mA, float **mB, float **mC, unsigned int m, unsigned int n, unsigned int p)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
 	
 	
 	if (row < m && col < p)  
    { 
    	mC[row][col] = 0;
	    for(int i = 0; i < n; i++){
	    	mC[row][col] += mA[row][i] * mB[i][col];
	    }
    }
}

void cuda(float *host_array_A, float *host_array_B, float *host_array_C)
{
	float **device_matrix_A = NULL;  
    float **host_matrix_A = NULL;  
    float *device_array_A = NULL;  
    //float *host_array_A = NULL;  
    hipError_t res;  
    int r, c;    
  
    res = hipMalloc((void**)(&device_matrix_A), M_*sizeof(float*));CHECK(res)  
    res = hipMalloc((void**)(&device_array_A), M_*N_*sizeof(float));CHECK(res)  
    host_matrix_A = (float**)malloc(M_*sizeof(float*));  
    //host_array_A = (float*)malloc(M_*N_*sizeof(float));  
  
    for (r = 0; r < M_; r++)  
    {  
        host_matrix_A[r] = device_array_A + r*N_;  
    }  

    res = hipMemcpy((void*)(device_matrix_A), (void*)(host_matrix_A), M_*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)  
    dim3 dimBlock(16,16);  
    dim3 dimGrid((N_+dimBlock.x-1)/(dimBlock.x), (M_+dimBlock.y-1)/(dimBlock.y));  
    InitMatrix<<<dimGrid, dimBlock>>>(device_matrix_A, M_, N_, 1);  
    res = hipMemcpy((void*)(host_array_A), (void*)(device_array_A), M_*N_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    float **device_matrix_B = NULL;  
    float **host_matrix_B = NULL;  
    float *device_array_B = NULL;  
    //float *host_array_B = NULL;  
  
    res = hipMalloc((void**)(&device_matrix_B), N_*sizeof(float*));CHECK(res)  
    res = hipMalloc((void**)(&device_array_B), N_*P_*sizeof(float));CHECK(res)  
    host_matrix_B = (float**)malloc(N_*sizeof(float*));  
    //host_array_B = (float*)malloc(N_*P_*sizeof(float));  
  
    for (r = 0; r < N_; r++)  
    {  
        host_matrix_B[r] = device_array_B + r*P_;  
    }  

    res = hipMemcpy((void*)(device_matrix_B), (void*)(host_matrix_B), N_*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)   
    InitMatrix<<<dimGrid, dimBlock>>>(device_matrix_B, N_, P_, 2);  
    res = hipMemcpy((void*)(host_array_B), (void*)(device_array_B), N_*P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    float **device_matrix_C = NULL;  
    float **host_matrix_C = NULL;  
    float *device_array_C = NULL;  
    //float *host_array_C = NULL;  

    res = hipMalloc((void**)(&device_matrix_C), M_*sizeof(float*));CHECK(res)  
    res = hipMalloc((void**)(&device_array_C), M_*P_*sizeof(float));CHECK(res)  
    host_matrix_C = (float**)malloc(M_*sizeof(float*));  
    //host_array_C = (float*)malloc(M_*P_*sizeof(float));  

    for (r = 0; r < M_; r++)  
    {  
        host_matrix_C[r] = device_array_C + r*P_;  
    } 

    res = hipMemcpy((void*)(device_matrix_C), (void*)(host_matrix_C), M_*sizeof(float*), hipMemcpyHostToDevice);CHECK(res) 
    Multiply<<<dimGrid, dimBlock>>>(device_matrix_A, device_matrix_B, device_matrix_C, M_, N_, P_);  
    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_*P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  
  
    hipFree((void*)device_matrix_A);  
    hipFree((void*)device_array_A);  
    hipFree((void*)device_matrix_B);  
    hipFree((void*)device_array_B);  
    hipFree((void*)device_matrix_C);  
    hipFree((void*)device_array_C); 
    free(host_matrix_A);  
    //free(host_array_A);  
    free(host_matrix_B);  
   	//free(host_array_B); 
    free(host_matrix_C);  
    //free(host_array_C); 
}  

void initArray(float *array, int len)
{
	for(int i = 0; i < len; i++){

	}
}
void sequential(float *host_array_A, float *host_array_B, float *host_array_C)
{
	for(int i = 0; i < M_; i++)
	{
		for(int j = 0; j < P_; j++)
		{	
			host_array_C[i * M_ + j] = 0;
			for(int k = 0; k < N_; k++)
			{
				host_array_C[i * M_ + j] += host_array_A[i * M_ + k] * host_array_B[k * N_ + j];
			}
		}
	}
}

void cublas(float *host_array_A, float *host_array_B, float *host_array_C)
{
	int lda = M_, ldb = N_, ldc = M_;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

 	printf("start\n");

    // Do the actual multiplication
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M_, N_, P_, alpha, host_array_A, lda, host_array_B, ldb, beta, host_array_C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}
int main(int argc, char **argv)  
{  
	int r, c;
	float *host_array_A = (float*)malloc(M_*N_*sizeof(float)); 
	float *host_array_B = (float*)malloc(P_*N_*sizeof(float));
	float *host_array_C_para = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_seq = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_cublas = (float*)malloc(M_*P_*sizeof(float));
	cuda(host_array_A, host_array_B, host_array_C_para);

	show(host_array_A, M_, N_);
	show(host_array_B, N_, P_);
	show(host_array_C_para, M_, P_);

	sequential(host_array_A, host_array_B, host_array_C_seq);

	show(host_array_C_seq, M_, P_);

    cublas(host_array_A, host_array_B, host_array_C_cublas);

    show(host_array_C_cublas, M_, P_);
    
	free(host_array_A); 
	free(host_array_B); 
	free(host_array_C_para); 
	free(host_array_C_seq); 
	free(host_array_C_cublas); 
    return 0;  
}  