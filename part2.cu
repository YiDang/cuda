#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <iostream>
#include <thrust/device_vector.h>

#define M_ 5
#define N_ 5
#define P_ 5

#define BLOCK_SIZE 32
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

#define show(matrix, lenm, lenn) for(int r = 0; r < lenm; r++){for (int c = 0; c < lenn; c++){printf("%.6f ", matrix[r*lenn+c]);}printf("\n");}printf("\n");

uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

__global__ void InitArray(double *a, unsigned int rows, unsigned int cols, int seed)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    
    if (row < rows && col < cols)  
    {  
        hiprand_init((row*cols + col) * seed, /* the seed controls the sequence of random values that are produced */
                  row, /* the sequence number is only important with multiple cores */
                  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                  &state);

        a[row * cols + col] = hiprand_uniform(&state);
        //a[row * cols + col] = row * cols + col;
    }  
}

__global__ void Multiply(double *arrayA, double *arrayB, double *arrayC)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  

 	if (row < M_ && col < P_)  
    { 	
    	#pragma unroll
	    for(int i = 0; i < N_; i++)
        {
	    	arrayC[row * P_ + col] += arrayA[row * N_ + i] * arrayB[i * P_ + col];
	    }
    }
}


//texture<double, 1, hipReadModeElementType> texA;
//texture<double, 1, hipReadModeElementType> texB;
texture<double,2,hipReadModeElementType> tex_A;
texture<double,2,hipReadModeElementType> tex_B;
__global__ void MultiplyTexture(double *arrayC)  
{  

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < M_ && y < P_)
    {
        double a = 0, b = 0;
        double temp_result = 0;
        //printf("idx:%d,%d,v:%f\n",y,x,a);
        for (int i = 0; i < N_; i++)
        {
            b = tex2D(tex_B, x+0.5f, i+0.5f);
            a = tex2D(tex_A, i+0.5f, y+0.5f);
            temp_result += a * b;
            //if(x == 0 && y == 1)printf("%f * %f, %f\n",a,b,temp_result);
        }
        arrayC[y * M_ + x] = temp_result;
    }
}

__global__ void Multi_SM(double *arrayA, double *arrayB, double *arrayC)  
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockDim.y * by + ty;  
    int col = blockDim.x * bx + tx;

    __shared__ double sharedM[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double sharedN[BLOCK_SIZE][BLOCK_SIZE];

    double v = 0.0;
    #pragma unroll
    for (int i = 0; i < (int)(ceil((double)N_/BLOCK_SIZE)); i++)
    {
        if (i*BLOCK_SIZE + tx < N_ && row < M_)
            sharedM[ty][tx] = arrayA[row*N_ + i*BLOCK_SIZE + tx];
        else
            sharedM[ty][tx] = 0.0;

        if (i*BLOCK_SIZE + ty < N_ && col < P_)
            sharedN[ty][tx] = arrayB[(i*BLOCK_SIZE + ty)*P_ + col];
        else
            sharedN[ty][tx] = 0.0;
        __syncthreads();
        #pragma unroll
        for(int j = 0; j < BLOCK_SIZE; j++)
            v += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (row < M_ && col < P_)
        arrayC[row*P_ + col] = v;
}

    
void cudaInit(double *host_array_A, int rows, int cols)
{
    hipError_t res;
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);  
    dim3 dimGrid((cols+dimBlock.x-1)/(dimBlock.x), (rows+dimBlock.y-1)/(dimBlock.y));

    double *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), rows * cols * sizeof(double));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), rows * cols * sizeof(double), hipMemcpyHostToDevice);CHECK(res)
    InitArray<<<dimGrid, dimBlock>>>(device_array_A, rows, cols, 1);
    res = hipMemcpy((void*)(host_array_A), (void*)(device_array_A), rows * cols * sizeof(double), hipMemcpyDeviceToHost);CHECK(res)  

    hipFree((void*)device_array_A);
} 

double cudaMul(double *host_array_A, double *host_array_B, double *host_array_C, int method)
{	
    hipError_t res;
     
    int maxd = std::max(P_ ,std::max(M_ , N_));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((M_ + dimBlock.x-1)/(dimBlock.x), (P_ + dimBlock.y-1)/(dimBlock.y));

    double *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), M_ * N_ * sizeof(double));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), M_ * N_ * sizeof(double), hipMemcpyHostToDevice);CHECK(res)

    double *device_array_B = NULL;
    res = hipMalloc((void**)(&device_array_B), N_ * P_ * sizeof(double));CHECK(res)
    res = hipMemcpy((void*)(device_array_B), (void*)(host_array_B), N_ * P_ * sizeof(double), hipMemcpyHostToDevice);CHECK(res)

    double *device_array_C = NULL;
    res = hipMalloc((void**)(&device_array_C), M_ * P_ * sizeof(double));CHECK(res)
    res = hipMemcpy((void*)(device_array_C), (void*)(host_array_C), M_ * P_ * sizeof(double), hipMemcpyHostToDevice);CHECK(res)

    double start = rdtsc();
    if(method == 0)
    {
    	Multiply<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C);
    }  
    else if(method == 1)
    {
    	Multi_SM<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C);
    }
    //else if(method == 2)
    //{
    //	hipChannelFormatDesc desc = hipCreateChannelDesc<double>(); 
    //	hipBindTexture(NULL, texA, device_array_A, desc, M_ * N_ * sizeof(double));
	//	hipBindTexture(NULL, texB, device_array_B, desc, N_ * P_ * sizeof(double));
	//	MultiplyTexture<<<dimGrid, dimBlock>>>(device_array_C);
    //}
    hipDeviceSynchronize();
    double end = rdtsc();

    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_ * P_*sizeof(double), hipMemcpyDeviceToHost);CHECK(res)

    hipFree((void*)device_array_A);
    hipFree((void*)device_array_B);
    hipFree((void*)device_array_C);
    
    return end - start;
}


double cudaMulTex(double *host_array_A, double *host_array_B, double *host_array_C)
{   
    hipError_t res;
     
    double *device_array_C = NULL;
    res = hipMalloc((void**)(&device_array_C), M_ * P_ * sizeof(double));CHECK(res)

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((M_ + dimBlock.x-1)/(dimBlock.x), (P_ + dimBlock.y-1)/(dimBlock.y));
    //..........................
    double (*d_a)[N_];
    double (*tmp1)[N_];

    tmp1 = (double (*)[N_])malloc(M_*N_*sizeof(double));

    for(int i = 0; i < M_ ; i++)
    {
        for(int j = 0; j < N_; j++)
        {
            tmp1[i][j] = host_array_A[i * N_ + j];
            //printf("%f ",tmp1[i][j]);
        }
        //printf("\n");
    }
    size_t pitch;
    hipMallocPitch((void**)&d_a, &pitch, N_*sizeof(double), M_);

    hipMemcpy2D(d_a,             // device destination                                   
                             pitch,           // device pitch (calculated above)                      
                             tmp1,               // src on host                                          
                             N_*sizeof(double), // pitch on src (no padding so just width of row)       
                             N_*sizeof(double), // width of data in bytes                               
                             M_,            // height of data                                       
                             hipMemcpyHostToDevice) ;
    hipBindTexture2D(NULL, tex_A, d_a, tex_A.channelDesc, N_, M_, pitch) ;
    tex_A.normalized = false;  // don't use normalized values                                           
    tex_A.filterMode = hipFilterModeLinear;
    tex_A.addressMode[0] = hipAddressModeClamp; // don't wrap around indices                           
    tex_A.addressMode[1] = hipAddressModeClamp;
    //..........................
    double (*d_b)[P_];
    double (*tmp2)[P_];

    tmp2 = (double (*)[P_])malloc(N_*P_*sizeof(double));

    for(int i = 0; i < N_ ; i++)
    {
        for(int j = 0; j < P_; j++)
        {
            tmp2[i][j] = host_array_A[i * P_ + j];
            //printf("%f ",tmp2[i][j]);
        }
        //printf("\n");
    }
    size_t pitch2;
    hipMallocPitch((void**)&d_b, &pitch2, P_*sizeof(double), N_);

    hipMemcpy2D(d_b,             // device destination                                   
                             pitch2,           // device pitch2 (calculated above)                      
                             tmp2,               // src on host                                          
                             P_*sizeof(double), // pitch2 on src (no padding so just width of row)       
                             P_*sizeof(double), // width of data in bytes                               
                             N_,            // height of data                                       
                             hipMemcpyHostToDevice) ;
    hipBindTexture2D(NULL, tex_B, d_b, tex_B.channelDesc, P_, N_, pitch2) ;
    tex_B.normalized = false;  // don't use normalized values                                           
    tex_B.filterMode = hipFilterModeLinear;
    tex_B.addressMode[0] = hipAddressModeClamp; // don't wrap around indices                           
    tex_B.addressMode[1] = hipAddressModeClamp;
    //..........................


    double start = rdtsc();

    MultiplyTexture<<<dimGrid, dimBlock>>>(device_array_C);

    //hipDeviceSynchronize();
    double end = rdtsc();

    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_ * P_*sizeof(double), hipMemcpyDeviceToHost);CHECK(res)

    free(tmp1);
    hipFree((void*)d_a);
    free(tmp2);
    hipFree((void*)d_b);
    hipFree((void*)device_array_C);

    
    return end - start;

}

double sequential(double *host_array_A, double *host_array_B, double *host_array_C)
{	
	double start = rdtsc();
	for(int i = 0; i < M_; i++)
	{
		for(int j = 0; j < P_; j++)
		{	
			host_array_C[i * P_ + j] = 0;
			for(int k = 0; k < N_; k++)
			{
				//printf("index%d\n", i * M_ + j);
				host_array_C[i * P_ + j] += host_array_A[i * N_ + k] * host_array_B[k * P_ + j];
				//printf("%2f,%2f,%2f,\n", host_array_A[i * N_ + k], host_array_B[k * P_ + j], host_array_C[i * M_ + j]);
			}
		}
	}
	double end = rdtsc();
	return end - start;
}

double cublas(double *host_array_A, double *host_array_B, double *host_array_C)
{

	thrust::host_vector<double> hvA(M_ * N_);
	thrust::host_vector<double> hvB(N_ * P_);
	thrust::host_vector<double> hvC(M_ * P_);
	for(int i = 0; i < M_ * N_; i++) 
	{
		hvA[i] = host_array_A[i];
	}
	for(int i = 0; i < P_ * N_; i++) 
	{
		hvB[i] = host_array_B[i];
	}
    
	thrust::device_vector<double> dvA = hvA;
	thrust::device_vector<double> dvB = hvB;
	thrust::device_vector<double> dvC(M_ * P_);

    int lda=N_ ,ldb=P_, ldc=P_;
    const double alpha = 1.0f;
    const double beta = 0.0f;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! CUBLAS initialization error\n";
    }
    
    // Do the actual multiplication
    double start = rdtsc();
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            P_, M_, N_, 
                            &alpha, 
                            thrust::raw_pointer_cast(&dvB[0]), ldb, 
                            thrust::raw_pointer_cast(&dvA[0]), lda, 
                            &beta, 
                            thrust::raw_pointer_cast(&dvC[0]), ldc);

    double end = rdtsc();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! kernel execution error.\n";
    }
    hvC = dvC;
    for(int i = 0; i < M_ * P_; i++) 
    {
    	host_array_C[i] = hvC[i];
    }

    // Destroy the handle
    hipblasDestroy(handle);

    return end - start;
}

int main(int argc, char **argv)  
{  
	double *host_array_A = (double*)malloc(M_*N_*sizeof(double)); 
	double *host_array_B = (double*)malloc(P_*N_*sizeof(double));
	double *host_array_C_seq = (double*)malloc(M_*P_*sizeof(double));
	double *host_array_C_cuda = (double*)malloc(M_*P_*sizeof(double));
	double *host_array_C_tile = (double*)malloc(M_*P_*sizeof(double));
	double *host_array_C_texture = (double*)malloc(M_*P_*sizeof(double));
	double *host_array_C_cublas = (double*)malloc(M_*P_*sizeof(double));

    int showma = 1, showdif = 1;
	double diff = 0;
    cudaInit(host_array_A, M_, N_);
	//show(host_array_A, M_, N_);
    cudaInit(host_array_B, N_, P_);
	//show(host_array_B, N_, P_);

	printf("cublas start\n");
    diff = 0;diff = cublas(host_array_A, host_array_B, host_array_C_cublas);
    if(showma) show(host_array_C_cublas, M_, P_);
    std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    printf("cuda start\n");
    diff = 0;diff = cudaMul(host_array_A, host_array_B, host_array_C_cuda, 0);
	if(showma) show(host_array_C_cuda, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;
    double error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_cuda[i];
    	error += tmp * tmp;
        if(tmp != 0 && showdif)
        {
            printf("cuda:%f",tmp);
        }
    }
    if(showdif)std::cout << "error:\t\t"<< error << std::endl << std::endl;

	printf("cuda tiled start\n");
    diff = 0;diff = cudaMul(host_array_A, host_array_B, host_array_C_tile, 1);
	if(showma) show(host_array_C_tile, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_tile[i];
    	error += tmp * tmp;
        if(tmp != 0 && showdif)
        {
            printf("tile:%f",tmp);
        }
    }
    if(showdif)std::cout << "error:\t\t"<< error << std::endl << std::endl;

    printf("cuda textured start\n");
    diff = 0;diff = cudaMulTex(host_array_A, host_array_B, host_array_C_texture);
	if(showma)show(host_array_C_texture, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_texture[i];
    	error += tmp * tmp;
        if(tmp != 0 && showdif)
        {
            printf("texture:%f ",tmp);
        }
    }
    if(showdif)std::cout << "error:\t\t"<< error << std::endl << std::endl;

    printf("seq start\n");
	diff = 0;diff = sequential(host_array_A, host_array_B, host_array_C_seq);
	if(showma) show(host_array_C_seq, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

   	error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_seq[i];
    	error += tmp * tmp;
        if(tmp != 0.0f && showdif)
        {
            printf("seq:%f,",tmp);
        }
    }
    if(showdif)std::cout << "error:\t\t"<< error << std::endl << std::endl;

	free(host_array_A); 
	free(host_array_B);  
	free(host_array_C_seq); 
	free(host_array_C_cuda); 
	free(host_array_C_tile); 
	free(host_array_C_cublas); 
	free(host_array_C_texture); 
	
    return 0;  
}  