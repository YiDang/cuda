#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define ROWS 32  
#define COLS 16  
#define MAX 100
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  
__global__ void Kerneltest(int **da, unsigned int rows, unsigned int cols)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    hiprand_init(0, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);
    if (row < rows && col < cols)  
    {  
        da[row][col] = hiprand(&state) % MAX;;  
    }  
}  
  
int main(int argc, char **argv)  
{  
    int **da = NULL;  
    int **ha = NULL;  
    int *dc = NULL;  
    int *hc = NULL;  
    hipError_t res;  
    int r, c;  
    bool is_right=true;  
  
    res = hipMalloc((void**)(&da), ROWS*sizeof(int*));CHECK(res)  
    res = hipMalloc((void**)(&dc), ROWS*COLS*sizeof(int));CHECK(res)  
    ha = (int**)malloc(ROWS*sizeof(int*));  
    hc = (int*)malloc(ROWS*COLS*sizeof(int));  
  
    for (r = 0; r < ROWS; r++)  
    {  
        ha[r] = dc + r*COLS;  
    }  
    res = hipMemcpy((void*)(da), (void*)(ha), ROWS*sizeof(int*), hipMemcpyHostToDevice);CHECK(res)  
    dim3 dimBlock(16,16);  
    dim3 dimGrid((COLS+dimBlock.x-1)/(dimBlock.x), (ROWS+dimBlock.y-1)/(dimBlock.y));  
    Kerneltest<<<dimGrid, dimBlock>>>(da, ROWS, COLS);  
    res = hipMemcpy((void*)(hc), (void*)(dc), ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost);CHECK(res)  
  
    for (r = 0; r < ROWS; r++)  
    {  
        for (c = 0; c < COLS; c++)  
        {  
            printf("%4d ", hc[r*COLS+c]);   
        }  
        printf("\n");  
    }  

    hipFree((void*)da);  
    hipFree((void*)dc);  
    free(ha);  
    free(hc);  

    return 0;  
}  