#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <iostream>
#include <thrust/device_vector.h>
#define M_ 1000 
#define N_ 1000
#define P_ 1000

#define BLOCK_SIZE 32
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

#define show(matrix, lenm, lenn) for(int r = 0; r < lenm; r++){for (int c = 0; c < lenn; c++){printf("%.6f ", matrix[r*lenn+c]);}printf("\n");}printf("\n");

uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

__global__ void InitArray(float *a, unsigned int rows, unsigned int cols, int seed)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    
    if (row < rows && col < cols)  
    {  
        hiprand_init((row*cols + col) * seed, /* the seed controls the sequence of random values that are produced */
                  row, /* the sequence number is only important with multiple cores */
                  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                  &state);

        a[row * cols + col] = hiprand_uniform(&state);
        //a[row * cols + col] = row * cols + col;
    }  
}

__global__ void Multiply(float *arrayA, float *arrayB, float *arrayC, unsigned int m, unsigned int n, unsigned int p)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  

 	if (row < M_ && col < P_)  
    { 	
    	#pragma unroll
	    for(int i = 0; i < N_; i++)
        {
	    	arrayC[row * P_ + col] += arrayA[row * N_ + i] * arrayB[i * P_ + col];
	    }
    }
}

__global__ void Multi_SM(float *arrayA, float *arrayB, float *arrayC, unsigned int m, unsigned int n, unsigned int p)  
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockDim.y * by + ty;  
    int col = blockDim.x * bx + tx;

    __shared__ float sharedM[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedN[BLOCK_SIZE][BLOCK_SIZE];

    float v = 0.0;
    #pragma unroll
    for (int i = 0; i < (int)(ceil((float)N_/BLOCK_SIZE)); i++)
    {
        if (i*BLOCK_SIZE + tx < N_ && row < M_)
            sharedM[ty][tx] = arrayA[row*N_ + i*BLOCK_SIZE + tx];
        else
            sharedM[ty][tx] = 0.0;

        if (i*BLOCK_SIZE + ty < N_ && col < P_)
            sharedN[ty][tx] = arrayB[(i*BLOCK_SIZE + ty)*P_ + col];
        else
            sharedN[ty][tx] = 0.0;
        __syncthreads();
        #pragma unroll
        for(int j = 0; j < BLOCK_SIZE; j++)
            v += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (row < M_ && col < P_)
        arrayC[row*P_ + col] = v;
}

    
void cudaInit(float *host_array_A, int rows, int cols)
{
    hipError_t res;
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);  
    dim3 dimGrid((cols+dimBlock.x-1)/(dimBlock.x), (rows+dimBlock.y-1)/(dimBlock.y));

    float *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), rows * cols * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), rows * cols * sizeof(float), hipMemcpyHostToDevice);CHECK(res)
    InitArray<<<dimGrid, dimBlock>>>(device_array_A, rows, cols, 1);
    res = hipMemcpy((void*)(host_array_A), (void*)(device_array_A), rows * cols * sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    hipFree((void*)device_array_A);
} 

double cudaMul(float *host_array_A, float *host_array_B, float *host_array_C, int method)
{	
	double start = rdtsc();
    hipError_t res;
     
    int maxd = std::max(P_ ,std::max(M_ , N_));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((maxd+ dimBlock.x-1)/(dimBlock.x), (maxd + dimBlock.y-1)/(dimBlock.y));

    float *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), M_ * N_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), M_ * N_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    float *device_array_B = NULL;
    res = hipMalloc((void**)(&device_array_B), N_ * P_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_B), (void*)(host_array_B), N_ * P_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    float *device_array_C = NULL;
    res = hipMalloc((void**)(&device_array_C), M_ * P_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_C), (void*)(host_array_C), M_ * P_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    
    if(method == 0)
    {
    	Multiply<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C, M_, N_, P_);
    }  
    else if(method == 1)
    {
    	Multi_SM<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C, M_, N_, P_);
    }
    

    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_ * P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)

    hipFree((void*)device_array_A);
    hipFree((void*)device_array_B);
    hipFree((void*)device_array_C);
    double end = rdtsc();
    return end - start;
}

double sequential(float *host_array_A, float *host_array_B, float *host_array_C)
{	
	double start = rdtsc();
	for(int i = 0; i < M_; i++)
	{
		for(int j = 0; j < P_; j++)
		{	
			host_array_C[i * P_ + j] = 0;
			for(int k = 0; k < N_; k++)
			{
				//printf("index%d\n", i * M_ + j);
				host_array_C[i * P_ + j] += host_array_A[i * N_ + k] * host_array_B[k * P_ + j];
				//printf("%2f,%2f,%2f,\n", host_array_A[i * N_ + k], host_array_B[k * P_ + j], host_array_C[i * M_ + j]);
			}
		}
	}
	double end = rdtsc();
	return end - start;
}

double cublas(float *host_array_A, float *host_array_B, float *host_array_C)
{
	thrust::host_vector<float> hvA(M_ * N_);
	thrust::host_vector<float> hvB(N_ * P_);
	thrust::host_vector<float> hvC(M_ * P_);
	for(int i = 0; i < M_ * N_; i++) 
	{
		hvA[i] = host_array_A[i];
	}
	for(int i = 0; i < P_ * N_; i++) 
	{
		hvB[i] = host_array_B[i];
	}
	thrust::device_vector<float> dvA = hvA;
	thrust::device_vector<float> dvB = hvB;
	thrust::device_vector<float> dvC(M_ * P_);

    int lda=N_ ,ldb=P_, ldc=P_;
    const float alpha = 1.0f;
    const float beta = 0.0f;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! CUBLAS initialization error\n";
    }
    double start = rdtsc();
    // Do the actual multiplication
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            P_, M_, N_, 
                            &alpha, 
                            thrust::raw_pointer_cast(&dvB[0]), ldb, 
                            thrust::raw_pointer_cast(&dvA[0]), lda, 
                            &beta, 
                            thrust::raw_pointer_cast(&dvC[0]), ldc);

    double end = rdtsc();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! kernel execution error.\n";
    }
    hvC = dvC;
    for(int i = 0; i < M_ * P_; i++) 
    {
    	host_array_C[i] = hvC[i];
    }

    // Destroy the handle
    hipblasDestroy(handle);

    return end - start;
}
int main(int argc, char **argv)  
{  
	float *host_array_A = (float*)malloc(M_*N_*sizeof(float)); 
	float *host_array_B = (float*)malloc(P_*N_*sizeof(float));
	float *host_array_C_seq = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_cuda = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_tile = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_cublas = (float*)malloc(M_*P_*sizeof(float));

	double diff = 0;
    cudaInit(host_array_A, M_, N_);
	//show(host_array_A, M_, N_);
    cudaInit(host_array_B, N_, P_);
	//show(host_array_B, N_, P_);

	printf("cublas start\n");
    diff = 0;diff = cublas(host_array_A, host_array_B, host_array_C_cublas);
    //show(host_array_C_cublas, M_, P_);
    std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    printf("cuda start\n");
    diff = 0;diff = cudaMul(host_array_A, host_array_B, host_array_C_cuda, 0);
	//show(host_array_C_cuda, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;
    double error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	int tmp = host_array_C_cublas - host_array_C_cuda;
    	error += tmp * tmp;
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;

	printf("cuda tiled start\n");
    diff = 0;diff = cudaMul(host_array_A, host_array_B, host_array_C_tile, 1);
	//show(host_array_C_tile, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	int tmp = host_array_C_cublas - host_array_C_tile;
    	error += tmp * tmp;
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;

    printf("seq start\n");
	diff = 0;diff = sequential(host_array_A, host_array_B, host_array_C_seq);
	//show(host_array_C_seq, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

   	error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	int tmp = host_array_C_cublas - host_array_C_seq;
    	error += tmp * tmp;
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;
  
	free(host_array_A); 
	free(host_array_B);  
	free(host_array_C_seq); 
	free(host_array_C_cuda); 
	free(host_array_C_tile); 
	free(host_array_C_cublas); 

    return 0;  
}  