#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <iostream>
#include <thrust/device_vector.h>
#define M_ 1000 
#define N_ 1000
#define P_ 1000

#define BLOCK_SIZE 32
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

#define show(matrix, lenm, lenn) for(int r = 0; r < lenm; r++){for (int c = 0; c < lenn; c++){printf("%.6f ", matrix[r*lenn+c]);}printf("\n");}printf("\n");

uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

__global__ void InitArray(float *a, unsigned int rows, unsigned int cols, int seed)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    
    if (row < rows && col < cols)  
    {  
        hiprand_init((row*cols + col) * seed, /* the seed controls the sequence of random values that are produced */
                  row, /* the sequence number is only important with multiple cores */
                  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                  &state);

        a[row][col] = hiprand_uniform(&state);
        //a[row * cols + col] = row * cols + col;
    }  
}

__global__ void Multiply(float *arrayA, float *arrayB, float *arrayC, unsigned int m, unsigned int n, unsigned int p)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  

 	if (row < m && col < p)  
    { 
	    for(int i = 0; i < n; i++)
        {
	    	arrayC[row * p + col] += arrayA[row * n + i] * arrayB[i * p + col];
	    }
    }
}

__global__ void Multi_SM(float *arrayA, float *arrayB, float *arrayC, unsigned int m, unsigned int n, unsigned int p)  
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockDim.y * by + ty;  
    int col = blockDim.x * bx + tx;

    __shared__ float sharedM[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedN[BLOCK_SIZE][BLOCK_SIZE];

    float v = 0.0;

    for (int i = 0; i < (int)(ceil((float)n/BLOCK_SIZE)); i++)
    {
        if (i*BLOCK_SIZE + tx < n && row < m)
            sharedM[ty][tx] = arrayA[row*n + i*BLOCK_SIZE + tx];
        else
            sharedM[ty][tx] = 0.0;

        if (i*BLOCK_SIZE + ty < n && col < p)
            sharedN[ty][tx] = arrayB[(i*BLOCK_SIZE + ty)*p + col];
        else
            sharedN[ty][tx] = 0.0;
        __syncthreads();

        for(int j = 0; j < BLOCK_SIZE; j++)
            v += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (row < m && col < p)
        arrayC[row*p + col] = v;
}

    
void cudaInit(float *host_array_A, int rows, int cols)
{
    hipError_t res;
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);  
    dim3 dimGrid((cols+dimBlock.x-1)/(dimBlock.x), (rows+dimBlock.y-1)/(dimBlock.y));

    float *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), rows * cols * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), rows * cols * sizeof(float), hipMemcpyHostToDevice);CHECK(res)
    InitArray<<<dimGrid, dimBlock>>>(device_array_A, rows, cols, 1);
    res = hipMemcpy((void*)(host_array_A), (void*)(device_array_A), rows * cols * sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    hipFree((void*)device_array_A);
} 

int cudaMul(float *host_array_A, float *host_array_B, float *host_array_C, int method)
{	
	
    hipError_t res;
     
    int maxd = std::max(P_ ,std::max(M_ , N_));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((maxd+ dimBlock.x-1)/(dimBlock.x), (maxd + dimBlock.y-1)/(dimBlock.y));

    float *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), M_ * N_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), M_ * N_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    float *device_array_B = NULL;
    res = hipMalloc((void**)(&device_array_B), N_ * P_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_B), (void*)(host_array_B), N_ * P_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    float *device_array_C = NULL;
    res = hipMalloc((void**)(&device_array_C), M_ * P_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_C), (void*)(host_array_C), M_ * P_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    int start = rdtsc();
    if(method == 0)
    {
    	Multiply<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C, M_, N_, P_);
    }  
    else if(method == 1)
    {
    	Multi_SM<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C, M_, N_, P_);
    }
    int end = rdtsc();

    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_ * P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)

    hipFree((void*)device_array_A);
    hipFree((void*)device_array_B);
    hipFree((void*)device_array_C);
    return end - start;
}

int sequential(float *host_array_A, float *host_array_B, float *host_array_C)
{	
	int start = rdtsc();
	for(int i = 0; i < M_; i++)
	{
		for(int j = 0; j < P_; j++)
		{	
			host_array_C[i * P_ + j] = 0;
			for(int k = 0; k < N_; k++)
			{
				//printf("index%d\n", i * M_ + j);
				host_array_C[i * P_ + j] += host_array_A[i * N_ + k] * host_array_B[k * P_ + j];
				//printf("%2f,%2f,%2f,\n", host_array_A[i * N_ + k], host_array_B[k * P_ + j], host_array_C[i * M_ + j]);
			}
		}
	}
	int end = rdtsc();
	return end - start;
}

int cublas(float *host_array_A, float *host_array_B, float *host_array_C)
{
	thrust::host_vector<float> hvA(M_ * N_);
	thrust::host_vector<float> hvB(N_ * P_);
	thrust::host_vector<float> hvC(M_ * P_);
	for(int i = 0; i < M_ * N_; i++) 
	{
		hvA[i] = host_array_A[i];
	}
	for(int i = 0; i < P_ * N_; i++) 
	{
		hvB[i] = host_array_B[i];
	}
	thrust::device_vector<float> dvA = hvA;
	thrust::device_vector<float> dvB = hvB;
	thrust::device_vector<float> dvC(M_ * P_);

    int lda=N_ ,ldb=P_, ldc=P_;
    const float alpha = 1.0f;
    const float beta = 0.0f;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! CUBLAS initialization error\n";
    }
    int start = rdtsc();
    // Do the actual multiplication
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            P_, M_, N_, 
                            &alpha, 
                            thrust::raw_pointer_cast(&dvB[0]), ldb, 
                            thrust::raw_pointer_cast(&dvA[0]), lda, 
                            &beta, 
                            thrust::raw_pointer_cast(&dvC[0]), ldc);

    int end = rdtsc();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! kernel execution error.\n";
    }
    hvC = dvC;
    for(int i = 0; i < M_ * P_; i++) 
    {
    	host_array_C[i] = hvC[i];
    }

    // Destroy the handle
    hipblasDestroy(handle);

    return end - start;
}
int main(int argc, char **argv)  
{  
	float *host_array_A = (float*)malloc(M_*N_*sizeof(float)); 
	float *host_array_B = (float*)malloc(P_*N_*sizeof(float));
	float *host_array_C = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_cublas = (float*)malloc(M_*P_*sizeof(float));

	int diff = 0;
    cudaInit(host_array_A, M_, N_);
	//show(host_array_A, M_, N_);
    cudaInit(host_array_B, N_, P_);
	//show(host_array_B, N_, P_);

    printf("cuda start\n");
    diff = cudaMul(host_array_A, host_array_B, host_array_C, 0);
	//show(host_array_C, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

	printf("cuda tiled start\n");
    diff = cudaMul(host_array_A, host_array_B, host_array_C, 1);
	//show(host_array_C, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    printf("seq start\n");
	diff = sequential(host_array_A, host_array_B, host_array_C);
	//show(host_array_C, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    printf("cublas start\n");
    diff = cublas(host_array_A, host_array_B, host_array_C_cublas);
    //show(host_array_C_cublas, M_, P_);
    std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;
    
	free(host_array_A); 
	free(host_array_B); 
	free(host_array_C); 
	free(host_array_C_cublas); 
    return 0;  
}  