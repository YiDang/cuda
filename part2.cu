#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <iostream>
#include <thrust/device_vector.h>

#define M_ 2
#define N_ 2
#define P_ 3

#define BLOCK_SIZE 32
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

#define show(matrix, lenm, lenn) for(int r = 0; r < lenm; r++){for (int c = 0; c < lenn; c++){printf("%.6f ", matrix[r*lenn+c]);}printf("\n");}printf("\n");

uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}

__global__ void InitArray(float *a, unsigned int rows, unsigned int cols, int seed)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    
    if (row < rows && col < cols)  
    {  
        hiprand_init((row*cols + col) * seed, /* the seed controls the sequence of random values that are produced */
                  row, /* the sequence number is only important with multiple cores */
                  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                  &state);

        //a[row * cols + col] = hiprand_uniform(&state);
        a[row * cols + col] = row * cols + col;
    }  
}

__global__ void Multiply(float *arrayA, float *arrayB, float *arrayC)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  

 	if (row < M_ && col < P_)  
    { 	
    	#pragma unroll
	    for(int i = 0; i < N_; i++)
        {
	    	arrayC[row * P_ + col] += arrayA[row * N_ + i] * arrayB[i * P_ + col];
	    }
    }
}


//texture<float, 1, hipReadModeElementType> texA;
//texture<float, 1, hipReadModeElementType> texB;
texture<float,2,hipReadModeElementType> tex_A;
texture<float,2,hipReadModeElementType> tex_B;
__global__ void MultiplyTexture(float *arrayC)  
{  

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < M_ && y < P_)
    {
        float a = 0, b = 0;
        //a = tex2D(tex_A, x+0.5f, y+0.5f);
        //b = tex2D(tex_B, y+0.5f, x+0.5f);
        //printf("%f * %f, xy:%d,%d\n",a,b,x,y);
        float temp_result = 0;
        //printf("idx:%d,%d,v:%f\n",y,x,a);
        for (int i = 0; i < N_; i++)
        {
            a = tex2D(tex_A, i+0.5f, x+0.5f);
            b = tex2D(tex_B, i+0.5f, y+0.5f);
            
            temp_result += a * b;
            printf("a%d,%d * b%d,%d  :%f * %f, %f, xy:%d,%d\n",i,x,i,y,a,b,temp_result,x,y);
        }
        arrayC[y * M_ + x] = temp_result;

    }
}

__global__ void Multi_SM(float *arrayA, float *arrayB, float *arrayC)  
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockDim.y * by + ty;  
    int col = blockDim.x * bx + tx;

    __shared__ float sharedM[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedN[BLOCK_SIZE][BLOCK_SIZE];

    float v = 0.0;
    #pragma unroll
    for (int i = 0; i < (int)(ceil((float)N_/BLOCK_SIZE)); i++)
    {
        if (i*BLOCK_SIZE + tx < N_ && row < M_)
            sharedM[ty][tx] = arrayA[row*N_ + i*BLOCK_SIZE + tx];
        else
            sharedM[ty][tx] = 0.0;

        if (i*BLOCK_SIZE + ty < N_ && col < P_)
            sharedN[ty][tx] = arrayB[(i*BLOCK_SIZE + ty)*P_ + col];
        else
            sharedN[ty][tx] = 0.0;
        __syncthreads();
        #pragma unroll
        for(int j = 0; j < BLOCK_SIZE; j++)
            v += sharedM[ty][j] * sharedN[j][tx];
        __syncthreads();
    }

    if (row < M_ && col < P_)
        arrayC[row*P_ + col] = v;
}

    
void cudaInit(float *host_array_A, int rows, int cols)
{
    hipError_t res;
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);  
    dim3 dimGrid((cols+dimBlock.x-1)/(dimBlock.x), (rows+dimBlock.y-1)/(dimBlock.y));

    float *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), rows * cols * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), rows * cols * sizeof(float), hipMemcpyHostToDevice);CHECK(res)
    InitArray<<<dimGrid, dimBlock>>>(device_array_A, rows, cols, 1);
    res = hipMemcpy((void*)(host_array_A), (void*)(device_array_A), rows * cols * sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    hipFree((void*)device_array_A);
} 

double cudaMul(float *host_array_A, float *host_array_B, float *host_array_C, int method)
{	
    hipError_t res;
     
    int maxd = std::max(P_ ,std::max(M_ , N_));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((M_ + dimBlock.x-1)/(dimBlock.x), (P_ + dimBlock.y-1)/(dimBlock.y));

    float *device_array_A = NULL;
    res = hipMalloc((void**)(&device_array_A), M_ * N_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_A), (void*)(host_array_A), M_ * N_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    float *device_array_B = NULL;
    res = hipMalloc((void**)(&device_array_B), N_ * P_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_B), (void*)(host_array_B), N_ * P_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    float *device_array_C = NULL;
    res = hipMalloc((void**)(&device_array_C), M_ * P_ * sizeof(float));CHECK(res)
    res = hipMemcpy((void*)(device_array_C), (void*)(host_array_C), M_ * P_ * sizeof(float), hipMemcpyHostToDevice);CHECK(res)

    double start = rdtsc();
    if(method == 0)
    {
    	Multiply<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C);
    }  
    else if(method == 1)
    {
    	Multi_SM<<<dimGrid, dimBlock>>>(device_array_A, device_array_B, device_array_C);
    }
    //else if(method == 2)
    //{
    //	hipChannelFormatDesc desc = hipCreateChannelDesc<float>(); 
    //	hipBindTexture(NULL, texA, device_array_A, desc, M_ * N_ * sizeof(float));
	//	hipBindTexture(NULL, texB, device_array_B, desc, N_ * P_ * sizeof(float));
	//	MultiplyTexture<<<dimGrid, dimBlock>>>(device_array_C);
    //}
    hipDeviceSynchronize();
    double end = rdtsc();

    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_ * P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)

    hipFree((void*)device_array_A);
    hipFree((void*)device_array_B);
    hipFree((void*)device_array_C);
    
    return end - start;
}


double cudaMulTex(float *host_array_A, float *host_array_B, float *host_array_C)
{   
    hipError_t res;
     
    float *device_array_C = NULL;
    res = hipMalloc((void**)(&device_array_C), M_ * P_ * sizeof(float));CHECK(res)
    int maxd = std::max(P_ ,std::max(M_ , N_));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((maxd + dimBlock.x-1)/(dimBlock.x), (maxd + dimBlock.y-1)/(dimBlock.y));
    //..........................
    float (*d_a)[N_];
    float (*tmp1)[N_];

    tmp1 = (float (*)[N_])malloc(M_*N_*sizeof(float));

    for(int i = 0; i < M_ ; i++)
    {
        for(int j = 0; j < N_; j++)
        {
            tmp1[i][j] = host_array_A[i * N_ + j];
            //printf("%f ",tmp1[i][j]);
        }
        //printf("\n");
    }
    size_t pitch;
    hipMallocPitch((void**)&d_a, &pitch, N_*sizeof(float), M_);

    hipMemcpy2D(d_a,             // device destination                                   
                             pitch,           // device pitch (calculated above)                      
                             tmp1,               // src on host                                          
                             N_*sizeof(float), // pitch on src (no padding so just width of row)       
                             N_*sizeof(float), // width of data in bytes                               
                             M_,            // height of data                                       
                             hipMemcpyHostToDevice) ;
    hipBindTexture2D(NULL, tex_A, d_a, tex_A.channelDesc, N_, M_, pitch) ;
    tex_A.normalized = false;  // don't use normalized values                                           
    tex_A.filterMode = hipFilterModeLinear;
    tex_A.addressMode[0] = hipAddressModeClamp; // don't wrap around indices                           
    tex_A.addressMode[1] = hipAddressModeClamp;
    //..........................
    float (*d_b)[P_];
    float (*tmp2)[P_];

    tmp2 = (float (*)[P_])malloc(N_*P_*sizeof(float));

    for(int i = 0; i < N_ ; i++)
    {
        for(int j = 0; j < P_; j++)
        {
            tmp2[i][j] = host_array_A[i * P_ + j];
            //printf("%f ",tmp2[i][j]);
        }
        //printf("\n");
    }
    size_t pitch2;
    hipMallocPitch((void**)&d_b, &pitch2, P_*sizeof(float), N_);

    hipMemcpy2D(d_b,             // device destination                                   
                             pitch2,           // device pitch2 (calculated above)                      
                             tmp2,               // src on host                                          
                             P_*sizeof(float), // pitch2 on src (no padding so just width of row)       
                             P_*sizeof(float), // width of data in bytes                               
                             N_,            // height of data                                       
                             hipMemcpyHostToDevice) ;
    hipBindTexture2D(NULL, tex_B, d_b, tex_B.channelDesc, P_, N_, pitch2) ;
    tex_B.normalized = false;  // don't use normalized values                                           
    tex_B.filterMode = hipFilterModeLinear;
    tex_B.addressMode[0] = hipAddressModeClamp; // don't wrap around indices                           
    tex_B.addressMode[1] = hipAddressModeClamp;
    //..........................


    double start = rdtsc();

    MultiplyTexture<<<dimGrid, dimBlock>>>(device_array_C);

    //hipDeviceSynchronize();
    double end = rdtsc();

    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_ * P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)

    free(tmp1);
    hipFree((void*)d_a);
    free(tmp2);
    hipFree((void*)d_b);
    hipFree((void*)device_array_C);

    
    return end - start;

}

double sequential(float *host_array_A, float *host_array_B, float *host_array_C)
{	
	double start = rdtsc();
	for(int i = 0; i < M_; i++)
	{
		for(int j = 0; j < P_; j++)
		{	
			host_array_C[i * P_ + j] = 0;
			for(int k = 0; k < N_; k++)
			{
				//printf("index%d\n", i * M_ + j);
				host_array_C[i * P_ + j] += host_array_A[i * N_ + k] * host_array_B[k * P_ + j];
				//printf("%2f,%2f,%2f,\n", host_array_A[i * N_ + k], host_array_B[k * P_ + j], host_array_C[i * M_ + j]);
			}
		}
	}
	double end = rdtsc();
	return end - start;
}

double cublas(float *host_array_A, float *host_array_B, float *host_array_C)
{

	thrust::host_vector<float> hvA(M_ * N_);
	thrust::host_vector<float> hvB(N_ * P_);
	thrust::host_vector<float> hvC(M_ * P_);
	for(int i = 0; i < M_ * N_; i++) 
	{
		hvA[i] = host_array_A[i];
	}
	for(int i = 0; i < P_ * N_; i++) 
	{
		hvB[i] = host_array_B[i];
	}
    
	thrust::device_vector<float> dvA = hvA;
	thrust::device_vector<float> dvB = hvB;
	thrust::device_vector<float> dvC(M_ * P_);

    int lda=N_ ,ldb=P_, ldc=P_;
    const float alpha = 1.0f;
    const float beta = 0.0f;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! CUBLAS initialization error\n";
    }
    
    // Do the actual multiplication
    double start = rdtsc();
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                            P_, M_, N_, 
                            &alpha, 
                            thrust::raw_pointer_cast(&dvB[0]), ldb, 
                            thrust::raw_pointer_cast(&dvA[0]), lda, 
                            &beta, 
                            thrust::raw_pointer_cast(&dvC[0]), ldc);

    double end = rdtsc();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "!!!! kernel execution error.\n";
    }
    hvC = dvC;
    for(int i = 0; i < M_ * P_; i++) 
    {
    	host_array_C[i] = hvC[i];
    }

    // Destroy the handle
    hipblasDestroy(handle);

    return end - start;
}

int main(int argc, char **argv)  
{  
	float *host_array_A = (float*)malloc(M_*N_*sizeof(float)); 
	float *host_array_B = (float*)malloc(P_*N_*sizeof(float));
	float *host_array_C_seq = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_cuda = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_tile = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_texture = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_cublas = (float*)malloc(M_*P_*sizeof(float));

    int showma = 0, showdif = 0;
	double diff = 0;
    cudaInit(host_array_A, M_, N_);
	//show(host_array_A, M_, N_);
    cudaInit(host_array_B, N_, P_);
	//show(host_array_B, N_, P_);
//----------------------------------------------------------------
	printf("cublas start\n");
    diff = 0;diff = cublas(host_array_A, host_array_B, host_array_C_cublas);
    if(showma) show(host_array_C_cublas, M_, P_);
    std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    printf("cuda start\n");
    diff = 0;diff = cudaMul(host_array_A, host_array_B, host_array_C_cuda, 0);
	if(showma) show(host_array_C_cuda, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;
    double error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_cuda[i];
    	error += tmp * tmp;
        if(tmp != 0 && showdif)
        {
            printf("cuda:%f",tmp);
        }
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;
//----------------------------------------------------------------
	printf("cuda tiled start\n");
    diff = 0;diff = cudaMul(host_array_A, host_array_B, host_array_C_tile, 1);
	if(showma) show(host_array_C_tile, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_tile[i];
    	error += tmp * tmp;
        if(tmp != 0 && showdif)
        {
            printf("tile:%f",tmp);
        }
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;
//----------------------------------------------------------------
    printf("cuda textured start\n");
    diff = 0;diff = cudaMulTex(host_array_A, host_array_B, host_array_C_texture);
	if(showma)show(host_array_C_texture, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

    error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_texture[i];
    	error += tmp * tmp;
        if(tmp != 0 && showdif)
        {
            printf("texture:%f ",tmp);
        }
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;
//----------------------------------------------------------------
    printf("seq start\n");
	diff = 0;diff = sequential(host_array_A, host_array_B, host_array_C_seq);
	if(showma) show(host_array_C_seq, M_, P_);
	std::cout << "Time million cycles:\t\t"
            << static_cast<double>(diff) / (1024 * 1024)
            << std::endl<< std::endl;

   	error = 0;
    for(int i = 0; i < M_ * N_; i++)
    {
    	double tmp = host_array_C_cublas[i] - host_array_C_seq[i];
    	error += tmp * tmp;
        if(tmp != 0.0f && showdif)
        {
            printf("seq:%f,",tmp);
        }
    }
    std::cout << "error:\t\t"<< error << std::endl << std::endl;
//----------------------------------------------------------------
	free(host_array_A); 
	free(host_array_B);  
	free(host_array_C_seq); 
	free(host_array_C_cuda); 
	free(host_array_C_tile); 
	free(host_array_C_cublas); 
	free(host_array_C_texture); 
	
    return 0;  
}  
