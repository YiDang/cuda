#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>  
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <iostream>
#include <thrust/device_vector.h>
#define M_ 2  
#define N_ 2 
#define P_ 2 

#define MAX 100
#define CHECK(res) if(res!=hipSuccess){exit(-1);}  

#define show(matrix, lenm, lenn) for(int r = 0; r < lenm; r++){for (int c = 0; c < lenn; c++){printf("%.6f ", matrix[r*lenn+c]);}printf("\n");}printf("\n");

hipError_t res;
__global__ void InitMatrix(float **m, unsigned int rows, unsigned int cols, int seed)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
    hiprandState_t state;
    hiprand_init((row*cols + col) * seed, /* the seed controls the sequence of random values that are produced */
              row, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &state);
    if (row < rows && col < cols)  
    {  
        //m[row][col] = hiprand_uniform(&state);
        m[row][col] = 1;
    }  
}  

__global__ void Multiply(float **mA, float **mB, float **mC, unsigned int m, unsigned int n, unsigned int p)  
{  
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;  
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;  
 	
 	
 	if (row < m && col < p)  
    { 
    	mC[row][col] = 0;
	    for(int i = 0; i < n; i++){
	    	mC[row][col] += mA[row][i] * mB[i][col];
	    }
    }
}

void cuda(float *host_array_A, float *host_array_B, float *host_array_C, float *device_array_A, float *device_array_B)
{
	float **device_matrix_A = NULL;  
    float **host_matrix_A = NULL;  
    //float *device_array_A = NULL;  
    //float *host_array_A = NULL;   
  
  
    res = hipMalloc((void**)(&device_matrix_A), M_*sizeof(float*));CHECK(res)  
    //res = hipMalloc((void**)(&device_array_A), M_*N_*sizeof(float));CHECK(res)  
    host_matrix_A = (float**)malloc(M_*sizeof(float*));  
    //host_array_A = (float*)malloc(M_*N_*sizeof(float));  
  
    for (int r = 0; r < M_; r++)  
    {  
        host_matrix_A[r] = device_array_A + r*N_;  
    }  

    res = hipMemcpy((void*)(device_matrix_A), (void*)(host_matrix_A), M_*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)  
    dim3 dimBlock(16,16);  
    dim3 dimGrid((N_+dimBlock.x-1)/(dimBlock.x), (M_+dimBlock.y-1)/(dimBlock.y));  
    InitMatrix<<<dimGrid, dimBlock>>>(device_matrix_A, M_, N_, 1);  
    res = hipMemcpy((void*)(host_array_A), (void*)(device_array_A), M_*N_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    float **device_matrix_B = NULL;  
    float **host_matrix_B = NULL;  
    //float *device_array_B = NULL;  
    //float *host_array_B = NULL;  
  
    res = hipMalloc((void**)(&device_matrix_B), N_*sizeof(float*));CHECK(res)  
    //res = hipMalloc((void**)(&device_array_B), N_*P_*sizeof(float));CHECK(res)  
    host_matrix_B = (float**)malloc(N_*sizeof(float*));  
    //host_array_B = (float*)malloc(N_*P_*sizeof(float));  
  
    for (int r = 0; r < N_; r++)  
    {  
        host_matrix_B[r] = device_array_B + r*P_;  
    }  

    res = hipMemcpy((void*)(device_matrix_B), (void*)(host_matrix_B), N_*sizeof(float*), hipMemcpyHostToDevice);CHECK(res)   
    InitMatrix<<<dimGrid, dimBlock>>>(device_matrix_B, N_, P_, 2);  
    res = hipMemcpy((void*)(host_array_B), (void*)(device_array_B), N_*P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  

    float **device_matrix_C = NULL;  
    float **host_matrix_C = NULL;  
    float *device_array_C = NULL;  
    //float *host_array_C = NULL;  

    res = hipMalloc((void**)(&device_matrix_C), M_*sizeof(float*));CHECK(res)  
    res = hipMalloc((void**)(&device_array_C), M_*P_*sizeof(float));CHECK(res)  
    host_matrix_C = (float**)malloc(M_*sizeof(float*));  
    //host_array_C = (float*)malloc(M_*P_*sizeof(float));  

    for (int r = 0; r < M_; r++)  
    {  
        host_matrix_C[r] = device_array_C + r*P_;  
    } 

    res = hipMemcpy((void*)(device_matrix_C), (void*)(host_matrix_C), M_*sizeof(float*), hipMemcpyHostToDevice);CHECK(res) 
    Multiply<<<dimGrid, dimBlock>>>(device_matrix_A, device_matrix_B, device_matrix_C, M_, N_, P_);  
    res = hipMemcpy((void*)(host_array_C), (void*)(device_array_C), M_*P_*sizeof(float), hipMemcpyDeviceToHost);CHECK(res)  
  
    hipFree((void*)device_matrix_A);  
    //hipFree((void*)device_array_A);  
    hipFree((void*)device_matrix_B);  
    //hipFree((void*)device_array_B);  
    hipFree((void*)device_matrix_C);  
    hipFree((void*)device_array_C); 
    free(host_matrix_A);  
    //free(host_array_A);  
    free(host_matrix_B);  
   	//free(host_array_B); 
    free(host_matrix_C);  
    //free(host_array_C); 
}  


void sequential(float *host_array_A, float *host_array_B, float *host_array_C)
{
	for(int i = 0; i < M_; i++)
	{
		for(int j = 0; j < P_; j++)
		{	
			host_array_C[i * M_ + j] = 0;
			for(int k = 0; k < N_; k++)
			{
				//printf("index%d\n", i * M_ + j);
				host_array_C[i * M_ + j] += host_array_A[i * N_ + k] * host_array_B[k * P_ + j];
				//printf("%2f,%2f,%2f,\n", host_array_A[i * N_ + k], host_array_B[k * P_ + j], host_array_C[i * M_ + j]);
			}
		}
	}
}

void cublas(float *device_array_A, float *device_array_B, float *host_array_C)
{
 	printf("start\n");


	for(int i = 0; i < M_ * N_; i++)
	{
		std::cout  << " ";
	}
	for(int i = 0; i < P_ * N_; i++)
	{
		std::cout<<device_array_A[i]<< " ";
	}
    // Do the actual multiplication

    int lda=N_ ,ldb=P_, ldc=P_;
	const float alpha = 1.0f;
	const float beta = 0.0f;
 
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
    	std::cerr << "!!!! CUBLAS initialization error\n";
  	}

    // Do the actual multiplication
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    						P_, M_, N_, 
    						&alpha, 
    						host_array_B, ldb, 
    						host_array_A, lda, 
    						&beta, 
    						host_array_C, ldc);
 	if (status != HIPBLAS_STATUS_SUCCESS) {
    	std::cerr << "!!!! kernel execution error.\n";
  	}

    // Destroy the handle
    hipblasDestroy(handle);

}
int main(int argc, char **argv)  
{  
	float *host_array_A = (float*)malloc(M_*N_*sizeof(float)); 
	float *device_array_A = NULL;
	res = hipMalloc((void**)(&device_array_A), M_*N_*sizeof(float));CHECK(res) 

	float *host_array_B = (float*)malloc(P_*N_*sizeof(float));
	float *device_array_B = NULL;
	res = hipMalloc((void**)(&device_array_B), N_*P_*sizeof(float));CHECK(res)  

	float *host_array_C_para = (float*)malloc(M_*P_*sizeof(float));
	float *host_array_C_seq = (float*)malloc(M_*P_*sizeof(float));
	float *device_array_C_cublas = NULL;
	res = hipMalloc((void**)(&device_array_C_cublas), M_*P_*sizeof(float));CHECK(res)  
	cuda(host_array_A, host_array_B, host_array_C_para, device_array_A, device_array_B);

	//show(host_array_A, M_, N_);
	//show(host_array_B, N_, P_);
	//show(host_array_C_para, M_, P_);

	sequential(host_array_A, host_array_B, host_array_C_seq);

	//show(host_array_C_seq, M_, P_);

    cublas(device_array_A, device_array_B, device_array_C_cublas);

    
    
	free(host_array_A); 
	free(host_array_B); 
	free(host_array_C_para); 
	free(host_array_C_seq); 
	hipFree((void*)device_array_A);  
	hipFree((void*)device_array_B);  
	hipFree((void*)device_array_C_cublas);  
    return 0;  
}  